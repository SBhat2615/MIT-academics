#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void program2(int *a, int* b, int* c, int n)
{
	int rid = threadIdx.x;
	for(int cid = 0; cid < n; cid++){
		for(int i = rid; i > 0; i--){
			c[rid*n + cid] *= c[rid*n + cid];
		}
	}
}


int main(void){
	int *a, m, n, i, j;
	int *d_a;

	printf("Enter the value of m: ");
	scanf("%d", &m);
	printf("Enter the value of n: ");
	scanf("%d", &n);

	int size = sizeof(int)*m*n;
	a = (int*)malloc(size);

	printf("Enter input matrix A:\n");
	for(i = 0; i < m*n; i++)
	scanf("%d", &a[i]);

	hipMalloc((void**)&d_a, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	// m threads
	program2<<<1, m>>>(d_a, n);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("Resultant matrix:\n");
	for(i = 0; i < m; i++){
		for(j = 0; j < n; j++)
			printf("%d\t", c[i*n+j]);
		printf("\n");
	}

	getchar();
	hipFree(d_a);
	return 0;
}

/*




*/