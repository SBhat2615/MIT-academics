#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void sparse1(int num_rows, float* data, int* col_index, int* row_ptr, float* x, float* y){
	
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < num_rows){
		// perform multiplication
	}
}

int main(void){
	int *d_a;
	int *data, *col_index, *row_ptr;
	int A[20][20], m, n;
	int B[m];

	printf("Enter size of matrix: ");
	scanf("%d%d", &m, &n);

	int size = m*n*sizeof(int);

	data = (int*)malloc(size);
	col_index = (int*)malloc(size);
	row_ptr = (int*)malloc((m+1)*sizeof(int));

	printf("Enter the elements for Matrix: ");
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++)
			scanf("%d", &A[i][j]);
	}

	printf("Enter the elements for Vector: ");
	for(int i = 0; i < m; i++){
		scanf("%d", &B[i]);
	}

	// Convert to Sparse
	int k = 0, l = 0;
	for(int i = 0; i < m; i++){
		int flag = 1;
		for(int j = 0; j < n; j++){
			if(A[i][j] != 0){
				data[k] = A[i][j];
				col_index[k] = j;
				if(flag){
					row_ptr[l++] = k;
					flag = 0;
				}
				k++;
			}
		}
	}

	/*
	printf("Sparse Matrix:\n");
	for(int i = 0; i < k; i++){
		printf("%d - %d\n", data[i], col_index[i]);
	}

	for(int i = 0; i < l; i++){
		printf("%d ", row_ptr[i]);
	}
	*/

	size1 = (k-1)*sizeof(int);
	size2 = m*sizeof(int);
	hipMalloc((void**)&d_a, size1);
	hipMalloc((void**)&d_b, size2);

	hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);
	// hipMemcpy for other kernel arguments(pointer args)

	// CALL KERNEL FUNCTION

	getchar();
	return 0;
}