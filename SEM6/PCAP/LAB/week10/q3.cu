#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__global__ void program3(int *a, int* b, int n)
{
	int rid = threadIdx.x;
	for(int cid = 0; cid < n; cid++){
		if(rid != n-1 || cid != n-1 || rid != 0 || cid != 0)
			// Change code to write 1's complement
			// b[rid*n + cid] = a[rid*n + cid];
		}
	}
}


int main(void){
	int *a, *b, m, n, i, j;
	int *d_a, *d_b;

	printf("Enter the value of m: ");
	scanf("%d", &m);
	printf("Enter the value of n: ");
	scanf("%d", &n);

	int size = sizeof(int)*m*n;
	a = (int*)malloc(size);
	b = (int*)malloc(size);

	printf("Enter input matrix A:\n");
	for(i = 0; i < m*n; i++)
	scanf("%d", &a[i]);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	// m threads
	program3<<<1, m>>>(d_a, d_b, n);

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	printf("Resultant matrix:\n");
	for(i = 0; i < m; i++){
		for(j = 0; j < n; j++)
			printf("%d\t", c[i*n+j]);
		printf("\n");
	}

	getchar();
	hipFree(d_a);
	return 0;
}

/*




*/