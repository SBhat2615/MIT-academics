#include <stdio.h>
#include "hip/hip_runtime.h"

#include <stdlib.h>

__global__ void add(int *a, int *b, int *c)
{
	*c = *a + *b;
}

__global__ void add1(int* a, int* b, int* c)
{
	int tid = blockIdx.x;
	c[tid] = a[tid] + b[tid];
}

__global__ void add2(int* a, int* b, int* c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

__global__ void question2(int* a, int* b, int* c, int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
		c[tid] = a[tid] + b[tid];
}

__global__ void question3(float* N, float* M, float* P, int width, int mask_width){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float Pvalue = 0;
	int startpoint = tid - (mask_width/2);
	for(int j = 0; j < mask_width; j++){
		if(startpoint+j >=0 && startpoint+j < width){
			Pvalue += N[startpoint+j] * M[j];
		}
	}
	P[tid] = Pvalue;                                       
}

__global__ void question4(int* a, int* b){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	b[tid] = sin(a[tid]);
}


int main(void) {


	// Solved, Question1, Question2

	/*
	int *d_a, *d_b, *d_c;
	
	int N = 6;
	int size = sizeof(int) * N;

	cudaMalloc((void **)&d_a, size);
	cudaMalloc((void **)&d_b, size);
	cudaMalloc((void **)&d_c, size);
	
	int a[N], b[N], c[N];

	printf("Enter input for A: ");
	for(int i = 0; i < N; i++){
		scanf("%d", &a[i]);
	}
	printf("Enter input for B: ");
	for(int i = 0; i < N; i++){
		scanf("%d", &b[i]);
	}
	
	cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
	cudaMemcpy(d_b, b, size, cudaMemcpyHostToDevice);
	
	// add<<<1,1>>>(d_a, d_b, d_c);
	// add1<<<N,1>>>(d_a, d_b, d_c);
	// add2<<<1,N>>>(d_a, d_b, d_c);

	dim3 dimGrid(ceil(N/256.0), 1, 1);
	dim3 dimBlock(256, 1, 1);
	question2<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N);

	cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);
	printf("Result: ");
	for(int i=0; i<N; i++)
		printf("%d ",c[i]);
	printf("\n");

	cudaFree(d_a);
	cudaFree(d_b);
	cudaFree(d_c);
	return 0;
	*/


	// Question3

	/*
	float *d_N, *d_M, *d_P;
	
	int l = 6;
	int size = sizeof(float) * l;

	int mask_width = 3;
	float M[] = {1.0,2.0,3.0};
	int maskSize = sizeof(float) * mask_width;

	cudaMalloc((void **)&d_N, size);
	cudaMalloc((void **)&d_M, maskSize);
	cudaMalloc((void **)&d_P, size);
	
	float N[l], P[l];
	printf("Enter input for array N: ");
	for(int i = 0; i < l; i++){
		scanf("%f", &N[i]);
	}
	
	cudaMemcpy(d_N, N, size, cudaMemcpyHostToDevice);
	cudaMemcpy(d_M, M, maskSize, cudaMemcpyHostToDevice);
	
	dim3 dimGrid(ceil(l/256.0), 1, 1); // l/256 blocks
	dim3 dimBlock(256, 1, 1);
	question3<<<dimGrid, dimBlock>>>(d_N, d_M, d_P, l, mask_width);

	cudaMemcpy(P, d_P, size, cudaMemcpyDeviceToHost);
	printf("Result: ");
	for(int i=0; i<l; i++)
		printf("%f ",P[i]);
	printf("\n");

	cudaFree(d_N);
	cudaFree(d_M);
	cudaFree(d_P);
	return 0;
	*/

	// Question 4

	int *d_a, *d_b;
	
	int N = 6;
	int size = sizeof(int) * N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	
	int a[N], b[N];

	printf("Enter angle in radians: ");
	for(int i = 0; i < N; i++){
		scanf("%d", &a[i]);
	}
	
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(N/256.0), 1, 1);
	dim3 dimBlock(256, 1, 1);
	question4<<<dimGrid, dimBlock>>>(d_a, d_b);

	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);

	printf("Sin values of given array: ");
	for(int i=0; i<N; i++)
		printf("%d ",b[i]);
	printf("\n");

	hipFree(d_a);
	hipFree(d_b);
	return 0;
	
}