#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void addA(int *a, int* b, int* c, int n)
{
	int rid = threadIdx.x;
	for(int cid = 0; cid < n; cid++){
		c[rid*n + cid] = a[rid*n + cid] + b[rid*n + cid];
	}
}

__global__ void addB(int *a, int* b, int* c, int m, int n)
{
	int cid = threadIdx.x;
	for(int rid = 0; rid < m; rid++){
		c[rid*n + cid] = a[rid*n + cid] + b[rid*n + cid];
	}
}

__global__ void addC(int *a, int* b, int* c)
{
	int tid = threadIdx.x;
	c[tid] = a[tid] + b[tid];
}

int main(void){
	int *a, *b, *c, m, n, i, j;
	int *d_a, *d_b, *d_c;

	printf("Enter the value of m: ");
	scanf("%d", &m);
	printf("Enter the value of n: ");
	scanf("%d", &n);

	int size = sizeof(int)*m*n;
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	printf("Enter input matrix A:\n");
	for(i = 0; i < m*n; i++)
	scanf("%d", &a[i]);
	printf("Enter input matrix B:\n");
	for(i = 0; i < m*n; i++)
	scanf("%d", &b[i]);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// m threads
	//addA<<<1, m>>>(d_a, d_b, d_c, n);

	// n threads
	//addB<<<1, n>>>(d_a, d_b, d_c, m, n);
	
	int threads = m*n;
	addC<<<1, threads>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
	printf("Resultant matrix:\n");
	for(i = 0; i < m; i++){
		for(j = 0; j < n; j++)
			printf("%d\t", c[i*n+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}

/*

a)
Enter the value of m: 3
Enter the value of n: 2
Enter input matrix A:
2 3 4 5 6 7
Enter input matrix B:
6 7 8 9 10 11
Resultant matrix:
8	10	
12	14	
16	18


b)
Enter the value of m: 2
Enter the value of n: 3
Enter input matrix A:
2 3 4 5 6 7
Enter input matrix B:
5 6 7 8 9 10
Resultant matrix:
7	9	11	
13	15	17


c)
Enter the value of m: 2
Enter the value of n: 4
Enter input matrix A:
2 4 5 6 7 9 3 1
Enter input matrix B:
8 3 9 2 5 6 7 3
Resultant matrix:
10	7	14	8	
12	15	10	4


*/