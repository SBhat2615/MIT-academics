#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void mulA(int *a, int* b, int* c, int wa, int wb)
{
	int ridA = threadIdx.x;
	int sum;
	for(int cidB = 0; cidB < wb; cidB++){
		sum = 0;
		for(int i = 0; i < wa; i++){
			sum += (a[ridA*wa + i] * b[i*wb + cidB]);
		}
		c[ridA*wb + cidB] = sum;
	}
}


int main(void){
	int *a, *b, *c, ha, wa, hb, wb, i, j;
	int *d_a, *d_b, *d_c;

	printf("Enter row and column for matrix A: ");
	scanf("%d%d", &ha, &wa);
	printf("Enter row and column for matrix B: ");
	scanf("%d%d", &hb, &wb);

	int size1 = sizeof(int)*ha*wa;
	int size2 = sizeof(int)*hb*wb;
	if(wa != hb){
		printf("Error in size\n");
		return -1;
	}
	int size3 = sizeof(int)*ha*wb;
	a = (int*)malloc(size1);
	b = (int*)malloc(size2);
	c = (int*)malloc(size3);

	printf("Enter input matrix A:\n");
	for(i = 0; i < ha*wa; i++)
	scanf("%d", &a[i]);
	printf("Enter input matrix B:\n");
	for(i = 0; i < hb*wb; i++)
	scanf("%d", &b[i]);

	hipMalloc((void**)&d_a, size1);
	hipMalloc((void**)&d_b, size2);
	hipMalloc((void**)&d_c, size3);

	hipMemcpy(d_a, a, size1, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size2, hipMemcpyHostToDevice);

	// ha threads
	mulA<<<1, ha>>>(d_a, d_b, d_c, wa, wb);
	
	// wb threads


	hipMemcpy(c, d_c, size3, hipMemcpyDeviceToHost);
	printf("Resultant matrix:\n");
	for(i = 0; i < ha; i++){
		for(j = 0; j < wb; j++)
			printf("%d\t", c[i*wb+j]);
		printf("\n");
	}
	getchar();
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}

/*

a)



b)



c)



*/